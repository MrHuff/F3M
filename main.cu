/*
 *  file name: matrix.cu
 *
 *  matrix.cu contains the code that realize some common used matrix operations in CUDA
 *
 *  this is a toy program for learning CUDA, some functions are reusable in other project
 *
 */
//#include <stdio.h>
#include "1_d_conv.cu"
#include "n_tree.cuh"
#include <hip/hip_runtime_api.h>
//#include "utils.h"
/*
*********************************************************************
function name: main
description: test and compare
parameters:
            none
return: none
*********************************************************************
*/


int main(int argc, char const *argv[]){

    /*
     * Implement "skip heuristic" for full interactions..., figure out how to calculate everything on cuda at the end, little interactions as possible. for each level of granularity...
     * */

    /*Pass array [i*n+j] just flattened matrix.... maybe not so much faster... inline stuff...
     * time building of laplace nodes and call to cuda kernel, time this using nvprof...
     * Wednesday? 4pm 16th of September.
     * debug the nans...why do I get them???
     *
     *
     * */

    /*
     * Evaluate performance and accuracy
     */
//    auto warmup_1 = std::chrono::high_resolution_clock::now();
//    res_ref = exact_ref * b_train;
//    auto end_warmup_1 = std::chrono::high_resolution_clock::now();
//    res = ffm_obj * b_train; //Horrendus complexity, needs to fixed now
//    auto end_warmup_2 = std::chrono::high_resolution_clock::now();

//    auto warmup_duration_1 = std::chrono::duration_cast<std::chrono::milliseconds>(end_warmup_1-warmup_1);
//    auto warmup_duration_2 = std::chrono::duration_cast<std::chrono::milliseconds>(end_warmup_2-end_warmup_1);
//    std::cout<<warmup_duration_1.count()<<std::endl;
//    std::cout<<warmup_duration_2.count()<<std::endl;
    int l_n = std::stoi(argv[1]);
    int n = std::stoi(argv[2]);
    float min_points = std::stof(argv[3]);
    int threshold = std::stoi(argv[4]);
    float a = std::stof(argv[5]);
    float b = std::stof(argv[6]);
    float ls = std::stof(argv[7]);
//    benchmark_1<3>(l_n,n,min_points,threshold,a,b,ls);
    benchmark_2<3>(l_n,n,min_points,threshold,a,b,ls);
    hipProfilerStop();
    hipDeviceReset();
    //chart out nodes, n, speed etc...
    return 0;
}
