/*
 *  file name: matrix.cu
 *
 *  matrix.cu contains the code that realize some common used matrix operations in CUDA
 *
 *  this is a toy program for learning CUDA, some functions are reusable in other project
 *
 */
#include <stdio.h>
#include "1_d_conv.cu"
#include "n_tree.cuh"
#include "utils.h"
/*
*********************************************************************
function name: main
description: test and compare
parameters:
            none
return: none
*********************************************************************
*/


int main(int argc, char const *argv[]){

    const std::string device_cuda = "cuda:0"; //officially retarded
    const std::string device_cpu = "cpu";
//    torch::manual_seed(0);

//    torch::Tensor X = read_csv<float>("X.csv",1000,3);
//    torch::Tensor b = read_csv<float>("V.csv",1000,2);
    torch::Tensor X = torch::rand({100,nd});
    torch::Tensor b = torch::randn({100,1});

    float ls = 3.0;
    float lambda = 1e-2;
    rbf_pointer<float> op,op_grad;
    hipMemcpyFromSymbol(&op, HIP_SYMBOL(rbf_pointer_func<float>), sizeof(rbf_pointer<float>)); //rbf_pointer_func,rbf_pointer_grad
    hipMemcpyFromSymbol(&op_grad, HIP_SYMBOL(rbf_pointer_grad<float>), sizeof(rbf_pointer<float>)); //rbf_pointer_func,rbf_pointer_grad

    FMM_obj<float> ffm_obj = FMM_obj<float>(X,X,ls,op,lambda,device_cuda);
    FMM_obj<float> ffm_obj_grad = FMM_obj<float>(X,X,ls,op_grad,lambda,device_cuda);

//    torch::Tensor output = ffm_obj_test*b;
//    exact_MV<float> exact_obj_test = exact_MV<float>(X,X,ls,op,lambda,device_cuda);
//    torch::Tensor output_ref = exact_obj_test*b;
    torch::Tensor b_inv,tridiag_matrix,log_det,trace;
//    std::tie(b_inv,tridiag_matrix) = CG(ffm_obj_test,b,(float) 1e-6,(int) 100,true);
    std::tie(log_det,trace) = trace_and_log_det_calc(ffm_obj,ffm_obj_grad,(int)10,(int)50,(float)1e-6);
    std::cout<<log_det<<std::endl;
    std::cout<<trace<<std::endl;

//    log_det = calculate_one_lanczos_triag(tridiag_matrix);
//    std::cout<<log_det<<std::endl;
//    std::cout<<b_inv<<std::endl;
//    std::cout<<tridiag_matrix<<std::endl;


//    X_data=X,X,ls,op,lambda,device_cuda
//    torch::Tensor output = FFM<float>(X,X,b,device_cuda,ls,op);
//    torch::Tensor output_ref = torch::zeros_like(output);
//    X = X.to(device_cuda);
//    b = b.to(device_cuda);
//    output_ref = output_ref.to(device_cuda);
//    rbf_call<float>(X, X, b, output_ref,ls,op, false);
//
//    std::cout<<output<<std::endl;
//    printf("--------------------------------------\n");
//    std::cout<<output_ref<<std::endl;
//    std::cout<<((output_ref-output)/output_ref).abs_().mean()<<std::endl;

}
