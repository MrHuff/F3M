#include "hip/hip_runtime.h"
//
// Created by rhu on 2020-07-05.
//

#pragma once
#include <torch/torch.h> //for n00bs like me, direct translation to python rofl
#include "1_d_conv.cu"
template <typename scalar_t>
__global__ void box_division(const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> X_data,
                             const torch::PackedTensorAccessor32<scalar_t,2,torch::RestrictPtrTraits> centers,
                             const torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> b,
                             const torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> old_indices,
                             torch::PackedTensorAccessor32<int,1,torch::RestrictPtrTraits> output,
                             int divide_num


){
    int i = blockIdx.x * blockDim.x + threadIdx.x; // current thread
    if (i>X_data.size(0)-1){return;}
    int old_ind = old_indices[i];
    int add = old_ind*divide_num;
    for (int k=0;k<nd;k++){
        output[i]+= (centers[old_ind][k]<=X_data[i][k])*b[k]+add;
    }
}



